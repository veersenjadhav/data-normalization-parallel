#include "hip/hip_runtime.h"


#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include <cmath>
#include <omp.h>
#include <stdlib.h>
#include <chrono>

#define ROWS 36634
#define FEATURES 14
#define DATASET "Dataset.csv"

using namespace std;
using namespace chrono;

__global__
void normalization(float *d_dataset, float *d_min, float *d_max, float *d_normalized)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < ROWS && j < FEATURES)
	{
		d_normalized[i * FEATURES + j] = (d_dataset[i * FEATURES + j] - d_min[j]) / (d_max[j] - d_min[j]);
	}
}

int main()
{
	string line, field;

	float* h_dataset = new float[ROWS * FEATURES];
	float* h_min = new float[FEATURES], *h_max = new float[FEATURES];
	float* h_normalized = new float[ROWS * FEATURES];

	float *d_dataset, *d_min, *d_max, *d_normalized;

	ifstream in(DATASET);

	int value = 0;
	while (getline(in, line))
	{
		stringstream ss(line);

		while (getline(ss, field, ','))
		{
			h_dataset[value] = (float)atof(field.c_str());
			value++;
		}
	}

	auto start = steady_clock::now();

#pragma omp parallel
	{
#pragma omp for
		for (int j = 0; j<FEATURES; j++)
		{
			double temp_min = h_dataset[j];
			for (int i = 1; i<ROWS; i++)
			{
				if (h_dataset[i * FEATURES + j] < temp_min)
				{
					temp_min = h_dataset[i * FEATURES + j];
				}
			}
			h_min[j] = temp_min;
		}

#pragma omp for
		for (int j = 0; j<FEATURES; j++)
		{
			double temp_max = h_dataset[j];
			for (int i = 0; i<ROWS; i++)
			{
				if (h_dataset[i * FEATURES + j] > temp_max)
				{
					temp_max = h_dataset[i * FEATURES + j];
				}
			}
			h_max[j] = temp_max;
		}
	}

	auto end = steady_clock::now();

	hipMalloc((void**)&d_dataset, ROWS*FEATURES * sizeof(float));
	hipMalloc((void**)&d_min, FEATURES * sizeof(float));
	hipMalloc((void**)&d_max, FEATURES * sizeof(float));
	hipMalloc((void**)&d_normalized, ROWS*FEATURES * sizeof(float));

	hipMemcpy(d_min, h_min, FEATURES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_max, h_max, FEATURES * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dataset, h_dataset, ROWS*FEATURES * sizeof(float), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(512, 2);
	dim3 blocksPerGrid(72, 7);

	auto start_CUDA = steady_clock::now();

	normalization<<<blocksPerGrid, threadsPerBlock >>>(d_dataset, d_min, d_max, d_normalized);
	hipDeviceSynchronize();

	auto end_CUDA = steady_clock::now();

	hipMemcpy(h_normalized, d_normalized, ROWS*FEATURES * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i<5; i++)
	{
		for (int j = 0; j<FEATURES; j++)
		{
			printf("%.8f \t", h_normalized[i * FEATURES + j]);
		}
		printf("\n");
	}

	cout << "\n Elapsed time in seconds : "
		<< chrono::duration_cast<chrono::microseconds>((end - start) + (end_CUDA - start_CUDA)).count()
		<< " microsec" << endl;

	hipFree(d_dataset); hipFree(d_min); hipFree(d_max); hipFree(d_normalized);

	delete[] h_dataset, h_min, h_max, h_normalized;

	return 0;
}